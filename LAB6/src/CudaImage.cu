#include "hip/hip_runtime.h"
/**
********************************************************************************
*
*   @file       CudaImage.cxx
*
*   @brief      Class to handle a greyscale image using Nvidia CUDA to speedup computations on GPU.
*
*   @version    1.0
*
*   @todo       Write here anything you did not implement.
*
*   @date       21/11/2019
*
*   @author     Franck Vidal
*   @author     YOUR NAME
*
*
********************************************************************************
*/


//******************************************************************************
//  Include
//******************************************************************************
#include <cmath> // Header file for abs and log
#include <limits>
#include <iostream>
#include <sstream>

#include <hip/hip_runtime.h> // Main cuda header for low-level driver API
#include <hip/hip_runtime_api.h> // Main cuda header for high-level runtime  programming in C
#include <hip/hip_runtime.h> // Main cuda header for high-level programming in C++
//#include <cutil_math.h>

#include "CudaImage.h"


int CudaImage::m_device_count = 0;


//--------------------------------------------------------
__global__ void  negativeFilterKernel(float* apOutputData,
                                      float* apInputData,
                                      float aMinValue,
                                      float aDynamicRange,
                                      int anImageSize)
//--------------------------------------------------------
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i < anImageSize)
        apOutputData[i] = aMinValue + aDynamicRange * (1.0 - (apInputData[i] - aMinValue) / aDynamicRange);
}


//----------------------------------------------------------
__global__ void  shiftScaleFilterKernel(float* apOutputData,
                                        float* apInputData,
                                        float aShiftValue,
                                        float aScaleValue,
                                        int anImageSize)
//----------------------------------------------------------
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i < anImageSize)
        apOutputData[i] = (apInputData[i] + aShiftValue) * aScaleValue;
}



//---------------------------------------------------
__global__ void  logFilterKernel(float* apOutputData,
                                 float* apInputData,
                                 int anImageSize)
//---------------------------------------------------
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i < anImageSize)
        apOutputData[i] = log(apInputData[i]);
}


//----------------------------------------------------------
__global__ void  flipHorizontallyKernel(float* apOutputData,
                                        float* apInputData,
                                        int anImageWidth,
                                        int anImageHeight)
//----------------------------------------------------------
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i < anImageWidth * anImageHeight)
    {
        // Retrieve the 2D index (x, y) from i

        // % is the "modulo operator", the remainder of i / m_width
        int x = i % anImageWidth;

        // where "/" is an integer division
        int y = i / anImageWidth;

        // Get the pixel
        apOutputData[i] = apInputData[y * anImageWidth + anImageWidth - x - 1];
    }
}


//--------------------------------------------------------
__global__ void  flipVerticallyKernel(float* apOutputData,
                                  float* apInputData,
                                  int anImageWidth,
                                  int anImageHeight)
//--------------------------------------------------------
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i < anImageWidth * anImageHeight)
    {
        // Retrieve the 2D index (x, y) from i

        // % is the "modulo operator", the remainder of i / m_width
        int x = i % anImageWidth;

        // where "/" is an integer division
        int y = i / anImageWidth;

        // Get the pixel
        apOutputData[i] = apInputData[(anImageHeight - y - 1) * anImageWidth + x];
    }
}


//--------------------------
CudaImage::CudaImage():
//--------------------------
        Image(),
        m_p_device_memory(0),
        m_computing_time(0),
        m_host_to_device_transfer(0),
        m_device_to_host_transfer(0),
        m_device_to_device_transfer(0)
//--------------------------
{
    getNumberOfDevices();
}


//-----------------------------------------
CudaImage::CudaImage(const Image& anImage):
//-----------------------------------------
        Image(anImage),
        m_p_device_memory(0),
        m_computing_time(0),
        m_host_to_device_transfer(0),
        m_device_to_host_transfer(0),
        m_device_to_device_transfer(0)
//-----------------------------------------
{
    getNumberOfDevices();
    loadHost2Device();
}


//---------------------------------------------
CudaImage::CudaImage(const CudaImage& anImage):
//---------------------------------------------
        Image(anImage),
        m_p_device_memory(0),
        m_computing_time(anImage.m_computing_time),
        m_host_to_device_transfer(anImage.m_host_to_device_transfer),
        m_device_to_host_transfer(anImage.m_device_to_host_transfer),
        m_device_to_device_transfer(anImage.m_device_to_device_transfer)
//---------------------------------------------
{
    getNumberOfDevices();
    loadDevice2Device(anImage.m_p_device_memory);
}


//-----------------------------------------
CudaImage::CudaImage(const float* apData,
                     unsigned int aWidth,
                     unsigned int aHeight):
//-----------------------------------------
        Image(apData, aWidth, aHeight),
        m_p_device_memory(0),
        m_computing_time(0),
        m_host_to_device_transfer(0),
        m_device_to_host_transfer(0),
        m_device_to_device_transfer(0)
//-----------------------------------------
{
    getNumberOfDevices();
    loadHost2Device();
}


//--------------------------------------------
CudaImage::CudaImage(unsigned int aWidth,
                     unsigned int aHeight,
                     float aDefaultValue):
//--------------------------------------------
        Image(aWidth, aHeight, aDefaultValue),
        m_p_device_memory(0),
        m_computing_time(0),
        m_host_to_device_transfer(0),
        m_device_to_host_transfer(0),
        m_device_to_device_transfer(0)
//--------------------------------------------
{
    getNumberOfDevices();
    loadHost2Device();
}


//---------------------
CudaImage::~CudaImage()
//---------------------
{
    // Release the memory
    destroy();
}


//------------------------------------
float CudaImage::getElasedTime() const
//------------------------------------
{
    return m_computing_time + m_host_to_device_transfer + m_device_to_host_transfer + m_device_to_device_transfer;
}



//-------------------------------------------------------
CudaImage& CudaImage::operator=(const CudaImage& anImage)
//-------------------------------------------------------
{
    if (this != &anImage)
    {
        Image::operator=(anImage);

        m_computing_time = anImage.m_computing_time;
        m_host_to_device_transfer = anImage.m_host_to_device_transfer;
        m_device_to_host_transfer = anImage.m_device_to_host_transfer;
        m_device_to_device_transfer = anImage.m_device_to_device_transfer;

        loadDevice2Device(anImage.m_p_device_memory);
    }

    return *this;
}


//-----------------------
void CudaImage::destroy()
//-----------------------
{
    Image::destroy();
    releaseDeviceMemory();
}


//-----------------------------------
void CudaImage::releaseDeviceMemory()
//-----------------------------------
{
    // Release memory on device
    if (m_p_device_memory)
    {
        hipFree(m_p_device_memory);
        checkCudaError(__FILE__, __FUNCTION__, __LINE__);

        m_p_device_memory = 0;
    }
}

//--------------------------------------------
void CudaImage::loadPGM(const char* aFileName)
//--------------------------------------------
{
    Image::loadPGM(aFileName);
    loadHost2Device();
}


//---------------------------------------------------
void CudaImage::loadPGM(const std::string& aFileName)
//---------------------------------------------------
{
    CudaImage::loadPGM(aFileName.c_str());
}


//--------------------------------------------
void CudaImage::savePGM(const char* aFileName)
//--------------------------------------------
{
    loadDevice2Host();
    Image::savePGM(aFileName);
}


//---------------------------------------------------
void CudaImage::savePGM(const std::string& aFileName)
//---------------------------------------------------
{
    CudaImage::savePGM(aFileName.c_str());
}


//----------------------------------------------
void CudaImage::loadASCII(const char* aFileName)
//----------------------------------------------
{
    Image::loadASCII(aFileName);
    loadHost2Device();
}


//-----------------------------------------------------
void CudaImage::loadASCII(const std::string& aFileName)
//-----------------------------------------------------
{
    CudaImage::loadASCII(aFileName.c_str());
}


//---------------------------------------------
void CudaImage::saveASCII(const char* aFileName)
//---------------------------------------------
{
    loadDevice2Host();
    Image::saveASCII(aFileName);
}


//-----------------------------------------------------
void CudaImage::saveASCII(const std::string& aFileName)
//-----------------------------------------------------
{
    CudaImage::saveASCII(aFileName.c_str());
}


//------------------------------
CudaImage CudaImage::operator!()
//------------------------------
{
    // Create an image of the right size
    CudaImage temp(getWidth(), getHeight(), 0.0);

    // Configure the kernel
    unsigned int image_size = m_width * m_height;
    int   block_size;
    int   min_grid_size;

    hipOccupancyMaxPotentialBlockSize(&min_grid_size, &block_size, negativeFilterKernel, 0, image_size);
    checkCudaError(__FILE__, __FUNCTION__, __LINE__);

    int dim_grid = image_size / block_size;
    if (image_size % block_size) ++dim_grid;

    // Create events
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    checkCudaError(__FILE__, __FUNCTION__, __LINE__);

    // Record start event
       hipEventRecord(start, 0);

       // Run the kernel
    float min_value = getMinValue();
    float range = getMaxValue() - min_value;
    negativeFilterKernel<<< dim_grid, block_size >>>(temp.m_p_device_memory,
            m_p_device_memory, min_value, range, image_size);
    hipDeviceSynchronize();
    checkCudaError(__FILE__, __FUNCTION__, __LINE__);


    // Record stop event
       hipEventRecord(stop, 0);
    checkCudaError(__FILE__, __FUNCTION__, __LINE__);

       // Compute the time in ms
       float duration;
    hipEventElapsedTime(&duration, start, stop);
    checkCudaError(__FILE__, __FUNCTION__, __LINE__);

    // Update the total time
    m_computing_time += duration;

    // Destroy the events
    hipEventDestroy(start);
    hipEventDestroy(stop);
    checkCudaError(__FILE__, __FUNCTION__, __LINE__);

    return temp;
}


//------------------------------------------------------
CudaImage CudaImage::shiftScaleFilter(float aShiftValue,
                                      float aScaleValue)
//------------------------------------------------------
{
    // Create an image of the right size
    CudaImage temp(getWidth(), getHeight(), 0.0);

    // Configure the kernel
    unsigned int image_size = m_width * m_height;
    int   block_size;
    int   min_grid_size;

    hipOccupancyMaxPotentialBlockSize(&min_grid_size, &block_size, shiftScaleFilterKernel, 0, image_size);
    checkCudaError(__FILE__, __FUNCTION__, __LINE__);

    int dim_grid = image_size / block_size;
    if (image_size % block_size) ++dim_grid;

    // Create events
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    checkCudaError(__FILE__, __FUNCTION__, __LINE__);

    // Record start event
       hipEventRecord(start, 0);

       // Run the kernel
    shiftScaleFilterKernel<<< dim_grid, block_size >>>(temp.m_p_device_memory,
            m_p_device_memory, aShiftValue, aScaleValue, image_size);
    hipDeviceSynchronize();
    checkCudaError(__FILE__, __FUNCTION__, __LINE__);


    // Record stop event
       hipEventRecord(stop, 0);
    checkCudaError(__FILE__, __FUNCTION__, __LINE__);

       // Compute the time in ms
       float duration;
    hipEventElapsedTime(&duration, start, stop);
    checkCudaError(__FILE__, __FUNCTION__, __LINE__);

    // Update the total time
    m_computing_time += duration;

    // Destroy the events
    hipEventDestroy(start);
    hipEventDestroy(stop);
    checkCudaError(__FILE__, __FUNCTION__, __LINE__);

    return temp;
}


//----------------------------------
CudaImage CudaImage::getNormalised()
//----------------------------------
{
    return shiftScaleFilter(-getMinValue(), 1.0 / (getMaxValue() - getMinValue()));
}


//----------------------------------
CudaImage CudaImage::getNormalized()
//----------------------------------
{
    return (getNormalised());
}


//------------------------------
CudaImage CudaImage::logFilter()
//------------------------------
{
    // Create an image of the right size
    CudaImage temp(getWidth(), getHeight(), 0.0);

    // Configure the kernel
    unsigned int image_size = m_width * m_height;
    int   block_size;
    int   min_grid_size;

    hipOccupancyMaxPotentialBlockSize(&min_grid_size, &block_size, logFilterKernel, 0, image_size);
    checkCudaError(__FILE__, __FUNCTION__, __LINE__);

    int dim_grid = image_size / block_size;
    if (image_size % block_size) ++dim_grid;

    // Create events
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    checkCudaError(__FILE__, __FUNCTION__, __LINE__);

    // Record start event
       hipEventRecord(start, 0);

       // Run the kernel
    logFilterKernel<<< dim_grid, block_size >>>(temp.m_p_device_memory,
            m_p_device_memory, image_size);
    hipDeviceSynchronize();
    checkCudaError(__FILE__, __FUNCTION__, __LINE__);


    // Record stop event
       hipEventRecord(stop, 0);
    checkCudaError(__FILE__, __FUNCTION__, __LINE__);

       // Compute the time in ms
       float duration;
    hipEventElapsedTime(&duration, start, stop);
    checkCudaError(__FILE__, __FUNCTION__, __LINE__);

    // Update the total time
    m_computing_time += duration;

    // Destroy the events
    hipEventDestroy(start);
    hipEventDestroy(stop);
    checkCudaError(__FILE__, __FUNCTION__, __LINE__);

    return temp;
}


//-------------------------------------
CudaImage CudaImage::flipHorizontally()
//-------------------------------------
{
    // Create an image of the right size
    CudaImage temp(getWidth(), getHeight(), 0.0);

    // Configure the kernel
    unsigned int image_size = m_width * m_height;
    int   block_size;
    int   min_grid_size;

    hipOccupancyMaxPotentialBlockSize(&min_grid_size, &block_size, flipHorizontallyKernel, 0, image_size);
    checkCudaError(__FILE__, __FUNCTION__, __LINE__);

    int dim_grid = image_size / block_size;
    if (image_size % block_size) ++dim_grid;

    // Create events
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    checkCudaError(__FILE__, __FUNCTION__, __LINE__);

    // Record start event
       hipEventRecord(start, 0);

       // Run the kernel
    flipHorizontallyKernel<<< dim_grid, block_size >>>(temp.m_p_device_memory,
            m_p_device_memory, m_width, m_height);
    hipDeviceSynchronize();
    checkCudaError(__FILE__, __FUNCTION__, __LINE__);


    // Record stop event
       hipEventRecord(stop, 0);
    checkCudaError(__FILE__, __FUNCTION__, __LINE__);

       // Compute the time in ms
       float duration;
    hipEventElapsedTime(&duration, start, stop);
    checkCudaError(__FILE__, __FUNCTION__, __LINE__);

    // Update the total time
    m_computing_time += duration;

    // Destroy the events
    hipEventDestroy(start);
    hipEventDestroy(stop);
    checkCudaError(__FILE__, __FUNCTION__, __LINE__);

    return temp;
}


//-----------------------------------
CudaImage CudaImage::flipVertically()
//-----------------------------------
{
    // Create an image of the right size
    CudaImage temp(getWidth(), getHeight(), 0.0);

    // Configure the kernel
    unsigned int image_size = m_width * m_height;
    int   block_size;
    int   min_grid_size;

    hipOccupancyMaxPotentialBlockSize(&min_grid_size, &block_size, flipVerticallyKernel, 0, image_size);
    checkCudaError(__FILE__, __FUNCTION__, __LINE__);

    int dim_grid = image_size / block_size;
    if (image_size % block_size) ++dim_grid;

    // Create events
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    checkCudaError(__FILE__, __FUNCTION__, __LINE__);

    // Record start event
       hipEventRecord(start, 0);

       // Run the kernel
    flipVerticallyKernel<<< dim_grid, block_size >>>(temp.m_p_device_memory,
            m_p_device_memory, m_width, m_height);
    hipDeviceSynchronize();
    checkCudaError(__FILE__, __FUNCTION__, __LINE__);


    // Record stop event
       hipEventRecord(stop, 0);
    checkCudaError(__FILE__, __FUNCTION__, __LINE__);

       // Compute the time in ms
       float duration;
    hipEventElapsedTime(&duration, start, stop);
    checkCudaError(__FILE__, __FUNCTION__, __LINE__);

    // Update the total time
    m_computing_time += duration;

    // Destroy the events
    hipEventDestroy(start);
    hipEventDestroy(stop);
    checkCudaError(__FILE__, __FUNCTION__, __LINE__);

    return temp;
}


//---------------------------------
int CudaImage::getNumberOfDevices()
//---------------------------------
{
    hipGetDeviceCount(&m_device_count);
    checkCudaError(__FILE__, __FUNCTION__, __LINE__);

    std::cerr << "Number of CUDA devices:\t" << m_device_count << std::endl;

    return m_device_count;
}


//-------------------------------------------------------
void CudaImage::checkCudaError(const char* aFileName,
                               const char* aFunctionName,
                               unsigned int aLineNumber)
//-------------------------------------------------------
{
    // Get the latest CUDA error.
    hipError_t error_code = hipGetLastError();

    // A CUDA error has occured.
    if (error_code != hipSuccess)
    {
        std::stringstream error_message;
        error_message << "CUDA error: " << hipGetErrorString(error_code) << std::endl;

        error_message << "\tin file: " << aFileName << std::endl;
        error_message << "\tin function: " << aFunctionName << std::endl;
        error_message << "\tat line: " << aLineNumber << std::endl;

    throw error_message.str();
    }
}


//-------------------------------
void CudaImage::loadHost2Device()
//-------------------------------
{
    // Release the device memory if needed
    releaseDeviceMemory();

    // Allocate the memory on device
    unsigned int image_size = m_width * m_height;
    unsigned int array_size = sizeof(float) * image_size;

    hipMalloc((void**) &m_p_device_memory, array_size);
    checkCudaError(__FILE__, __FUNCTION__, __LINE__);

    // Create events
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    checkCudaError(__FILE__, __FUNCTION__, __LINE__);

    // Record start event
       hipEventRecord(start, 0);

       // Copy host memory to device memory
    hipMemcpy(m_p_device_memory, &m_p_image[0], array_size, hipMemcpyHostToDevice);

    // Record stop event
       hipEventRecord(stop, 0);
    checkCudaError(__FILE__, __FUNCTION__, __LINE__);

       // Compute the time in ms
       float duration;
    hipEventElapsedTime(&duration, start, stop);
    checkCudaError(__FILE__, __FUNCTION__, __LINE__);

    // Update the total time
    m_host_to_device_transfer += duration;

    // Destroy the events
    hipEventDestroy(start);
    hipEventDestroy(stop);
    checkCudaError(__FILE__, __FUNCTION__, __LINE__);
}


//-------------------------------
void CudaImage::loadDevice2Host()
//-------------------------------
{
    unsigned int image_size = m_width * m_height;
    unsigned int array_size = sizeof(float) * image_size;

    // Create events
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    checkCudaError(__FILE__, __FUNCTION__, __LINE__);

    // Record start event
       hipEventRecord(start, 0);

       // Copy device memory to host memory
    hipMemcpy(&m_p_image[0], m_p_device_memory, array_size, hipMemcpyDeviceToHost);

    // Record stop event
       hipEventRecord(stop, 0);
    checkCudaError(__FILE__, __FUNCTION__, __LINE__);

       // Compute the time in ms
       float duration;
    hipEventElapsedTime(&duration, start, stop);
    checkCudaError(__FILE__, __FUNCTION__, __LINE__);

    // Update the total time
    m_device_to_host_transfer += duration;

    // Destroy the events
    hipEventDestroy(start);
    hipEventDestroy(stop);
    checkCudaError(__FILE__, __FUNCTION__, __LINE__);
}


//-------------------------------------------------------------
void CudaImage::loadDevice2Device(const float* apImageOnDevice)
//-------------------------------------------------------------
{
    // Release the device memory if needed
    releaseDeviceMemory();

    // Allocate the memory on device
    unsigned int image_size = m_width * m_height;
    unsigned int array_size = sizeof(float) * image_size;

    hipMalloc((void**) &m_p_device_memory, array_size);
    checkCudaError(__FILE__, __FUNCTION__, __LINE__);

    // Create events
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    checkCudaError(__FILE__, __FUNCTION__, __LINE__);

    // Record start event
       hipEventRecord(start, 0);

       // Copy device memory to device memory
    hipMemcpy(m_p_device_memory, apImageOnDevice, array_size, hipMemcpyDeviceToDevice);

    // Record stop event
       hipEventRecord(stop, 0);
    checkCudaError(__FILE__, __FUNCTION__, __LINE__);

       // Compute the time in ms
       float duration;
    hipEventElapsedTime(&duration, start, stop);
    checkCudaError(__FILE__, __FUNCTION__, __LINE__);

    // Update the total time
    m_device_to_device_transfer += duration;

    // Destroy the events
    hipEventDestroy(start);
    hipEventDestroy(stop);
    checkCudaError(__FILE__, __FUNCTION__, __LINE__);
}
