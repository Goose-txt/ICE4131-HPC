#include "hip/hip_runtime.h"
/**
********************************************************************************
*
*   @file       CudaImage.cxx
*
*   @brief      Class to handle a greyscale image using Nvidia CUDA to speedup computations on GPU.
*
*   @version    1.0
*
*   @todo       Write here anything you did not implement.
*
*   @date       21/11/2019
*
*   @author     Franck Vidal
*   @author     YOUR NAME
*
*
********************************************************************************
*/


//******************************************************************************
//  Include
//******************************************************************************
#include <cmath> // Header file for abs and log
#include <limits>
#include <iostream>
#include <sstream>

#include <hip/hip_runtime.h> // Main cuda header for low-level driver API
#include <hip/hip_runtime_api.h> // Main cuda header for high-level runtime  programming in C
#include <hip/hip_runtime.h> // Main cuda header for high-level programming in C++
//#include <cutil_math.h>

#include "CudaImage.h"


int CudaImage::m_device_count = 0;


__global__ void  logFilterKernel(float* apOutputData, float* apInputData)
{

}


//---------------------
CudaImage::CudaImage():
//---------------------
        Image()
//---------------------
{
    getNumberOfDevices();
}


//-----------------------------------------
CudaImage::CudaImage(const Image& anImage):
//-----------------------------------------
        Image(anImage)
//-----------------------------------------
{
    getNumberOfDevices();
}


//-----------------------------------------
CudaImage::CudaImage(const float* apData,
                     unsigned int aWidth,
                     unsigned int aHeight):
//-----------------------------------------
        Image(apData, aWidth, aHeight)
//-----------------------------------------
{
    getNumberOfDevices();
}


//-------------------------------------------
CudaImage::CudaImage(unsigned int aWidth,
                     unsigned int aHeight,
                     float aDefaultValue):
//-------------------------------------------
        Image(aWidth, aHeight, aDefaultValue)
//-------------------------------------------
{
    getNumberOfDevices();
}


//-------------------------------------------
void CudaImage::savePGM(const char* aFileName)
//-------------------------------------------
{
    loadDevice2Host();
    Image::savePGM(aFileName);
}


//---------------------------------------------
void CudaImage::saveASCII(const char* aFileName)
//---------------------------------------------
{
    loadDevice2Host();
    Image::saveASCII(aFileName);
}


//------------------------------------
CudaImage CudaImage::operator!() const
//------------------------------------
{
    // Create an image of the right size
    CudaImage temp(getWidth(), getHeight(), 0.0);

    //float min_value(getMinValue());
    //float max_value(getMaxValue());
    //float range(max_value - min_value);

    // There is no device
    if (!m_device_count)
    {
        // Use serial code
        Image::operator!();
    }

    loadHost2Device();

/*
    // Process every pixel of the sub-image
    for (unsigned int i = pixel_start_id; i <= pixel_end_id; ++i)
    {
        // Take care to preserve the dynamic of the image
        temp[i] = min_value + range * (1.0 - (m_p_image[i] - min_value) / range);
    }
*/

    return temp;
}


//----------------------------------------------------------
CudaImage CudaImage::shiftScaleFilter(float aShiftValue,
                                    float aScaleValue) const
//----------------------------------------------------------
{
    // Create an image of the right size
    CudaImage temp(getWidth(), getHeight(), 0.0);
/*
    // Get the work load
    unsigned int pixel_start_id = 0;
    unsigned int pixel_end_id = 0;
    workload(m_width * m_height, pixel_start_id, pixel_end_id);

    // Process every pixel of the sub-image
    for (unsigned int i = pixel_start_id; i <= pixel_end_id; ++i)
    {
        // Apply the shilft/scale filter
        temp[i] = (m_p_image[i] + aShiftValue) * aScaleValue;
    }

    // Get the process' rank
    int rank;
    Cuda_Comm_rank(Cuda_COMM_WORLD, &rank);

    // Master gather results from all the processes
    if (rank == ROOT)
    {
        int world_size;
        Cuda_Comm_size(Cuda_COMM_WORLD, &world_size);

        for (int i = 1; i < world_size; ++i)
        {
            int pixel_start_id;
            int pixel_end_id;

            Cuda_Status status;

            checkCudaError(Cuda_Recv(&pixel_start_id, 1, Cuda_INT, i, 0, Cuda_COMM_WORLD, &status));
            checkCudaError(Cuda_Recv(&pixel_end_id, 1, Cuda_INT, i, 1, Cuda_COMM_WORLD, &status));
            checkCudaError(Cuda_Recv(&temp[pixel_start_id], pixel_end_id - pixel_start_id + 1, Cuda_FLOAT, i, 2, Cuda_COMM_WORLD, &status));
        }
    }
    // Other processes send the data to the master
    else
    {

        checkCudaError(Cuda_Send(&pixel_start_id, 1, Cuda_INT, ROOT, 0, Cuda_COMM_WORLD));
        checkCudaError(Cuda_Send(&pixel_end_id,   1, Cuda_INT, ROOT, 1, Cuda_COMM_WORLD));
        checkCudaError(Cuda_Send(&temp[pixel_start_id], pixel_end_id - pixel_start_id + 1, Cuda_FLOAT, ROOT, 2, Cuda_COMM_WORLD));
    }
*/
    return temp;
}


//----------------------------------
CudaImage CudaImage::logFilter() const
//----------------------------------
{
    // Create an image of the right size
    CudaImage temp(getWidth(), getHeight(), 0.0);

	// Allocate the memory on device
    float* p_device_input_image_data  = 0;
    float* p_device_output_image_data = 0;

    unsigned int array_size = sizeof(float) * m_width * m_height;

    hipMalloc((void**) &p_device_input_image_data, array_size);
    hipMalloc((void**) &p_device_output_image_data, array_size);
	checkCudaError(__FILE__, __FUNCTION__, __LINE__);

    // Copy host memory to device memory
	hipMemcpy(p_device_input_image_data, &m_p_image[0], array_size, hipMemcpyHostToDevice);

    // Configure the kernel
	int   blockSize;
    int   minGridSize;

    // Get the work load
    hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, logFilterKernel, 0, m_width * m_height);
    checkCudaError(__FILE__, __FUNCTION__, __LINE__);


    // Note that 256 x 16 = 4096, i.e. the number of elements in the arrays.

	// Run the kernel
	logFilterKernel<<< minGridSize, blockSize >>>(p_device_output_image_data,
		p_device_input_image_data);
	hipDeviceSynchronize();
	checkCudaError(__FILE__, __FUNCTION__, __LINE__);

    // Copy host memory to device memory
    hipMemcpy(p_device_output_image_data, &(temp.m_p_image[0]), array_size, hipMemcpyDeviceToHost);

/*
    // Get the work load
    unsigned int pixel_start_id = 0;
    unsigned int pixel_end_id = 0;
    workload(m_width * m_height, pixel_start_id, pixel_end_id);

    // Process every pixel of the sub-image
    for (unsigned int i = pixel_start_id; i <= pixel_end_id; ++i)
    {
        // Apply the log filter
        temp[i] = log(m_p_image[i]);
    }

    // Get the process' rank
    int rank;
    Cuda_Comm_rank(Cuda_COMM_WORLD, &rank);

    // Master gather results from all the processes
    if (rank == ROOT)
    {
        int world_size;
        Cuda_Comm_size(Cuda_COMM_WORLD, &world_size);

        for (int i = 1; i < world_size; ++i)
        {
            int pixel_start_id;
            int pixel_end_id;

            Cuda_Status status;

            checkCudaError(Cuda_Recv(&pixel_start_id, 1, Cuda_INT, i, 0, Cuda_COMM_WORLD, &status));
            checkCudaError(Cuda_Recv(&pixel_end_id, 1, Cuda_INT, i, 1, Cuda_COMM_WORLD, &status));
            checkCudaError(Cuda_Recv(&temp[pixel_start_id], pixel_end_id - pixel_start_id + 1, Cuda_FLOAT, i, 2, Cuda_COMM_WORLD, &status));
        }
    }
    // Other processes send the data to the master
    else
    {
        checkCudaError(Cuda_Send(&pixel_start_id, 1, Cuda_INT, ROOT, 0, Cuda_COMM_WORLD));
        checkCudaError(Cuda_Send(&pixel_end_id,   1, Cuda_INT, ROOT, 1, Cuda_COMM_WORLD));
        checkCudaError(Cuda_Send(&temp[pixel_start_id], pixel_end_id - pixel_start_id + 1, Cuda_FLOAT, ROOT, 2, Cuda_COMM_WORLD));
    }
*/
    return temp;
}


//-----------------------------------------
CudaImage CudaImage::flipHorizontally() const
//-----------------------------------------
{
    // Create an image of the right size
    CudaImage temp(getWidth(), getHeight(), 0.0);
/*
    // Get the work load
    unsigned int pixel_start_id = 0;
    unsigned int pixel_end_id = 0;
    workload(m_width * m_height, pixel_start_id, pixel_end_id);

    // Process every pixel of the sub-image
    for (unsigned int i = pixel_start_id; i <= pixel_end_id; ++i)
    {
        // Retrieve the 2D index (x, y) from i

        // % is the "modulo operator", the remainder of i / m_width
        int x = i % m_width;

        // where "/" is an integer division
        int y = i / m_width;

        // Get the pixel
        temp(x, y) = getPixel(m_width - x - 1, y);
    }

    // Get the process' rank
    int rank;
    Cuda_Comm_rank(Cuda_COMM_WORLD, &rank);

    // Master gather results from all the processes
    if (rank == ROOT)
    {
        int world_size;
        Cuda_Comm_size(Cuda_COMM_WORLD, &world_size);

        for (int i = 1; i < world_size; ++i)
        {
            int pixel_start_id;
            int pixel_end_id;

            Cuda_Status status;

            checkCudaError(Cuda_Recv(&pixel_start_id, 1, Cuda_INT, i, 0, Cuda_COMM_WORLD, &status));
            checkCudaError(Cuda_Recv(&pixel_end_id, 1, Cuda_INT, i, 1, Cuda_COMM_WORLD, &status));
            checkCudaError(Cuda_Recv(&temp[pixel_start_id], pixel_end_id - pixel_start_id + 1, Cuda_FLOAT, i, 2, Cuda_COMM_WORLD, &status));
        }
    }
    // Other processes send the data to the master
    else
    {
        checkCudaError(Cuda_Send(&pixel_start_id, 1, Cuda_INT, ROOT, 0, Cuda_COMM_WORLD));
        checkCudaError(Cuda_Send(&pixel_end_id,   1, Cuda_INT, ROOT, 1, Cuda_COMM_WORLD));
        checkCudaError(Cuda_Send(&temp[pixel_start_id], pixel_end_id - pixel_start_id + 1, Cuda_FLOAT, ROOT, 2, Cuda_COMM_WORLD));
    }
*/
    return temp;
}


//---------------------------------------
CudaImage CudaImage::flipVertically() const
//---------------------------------------
{
    // Create an image of the right size
    CudaImage temp(getWidth(), getHeight(), 0.0);
/*
    // Get the work load
    unsigned int pixel_start_id = 0;
    unsigned int pixel_end_id = 0;
    workload(m_width * m_height, pixel_start_id, pixel_end_id);

    // Process every pixel of the sub-image
    for (unsigned int i = pixel_start_id; i <= pixel_end_id; ++i)
    {
        // Retrieve the 2D index (x, y) from i

        // % is the "modulo operator", the remainder of i / m_width
        int x = i % m_width;

        // where "/" is an integer division
        int y = i / m_width;

        // Get the pixel
        temp(x, y) = getPixel(x, m_height - y - 1);
    }

    // Get the process' rank
    int rank;
    Cuda_Comm_rank(Cuda_COMM_WORLD, &rank);

    // Master gather results from all the processes
    if (rank == ROOT)
    {
        int world_size;
        Cuda_Comm_size(Cuda_COMM_WORLD, &world_size);

        for (int i = 1; i < world_size; ++i)
        {
            int pixel_start_id;
            int pixel_end_id;

            Cuda_Status status;

            checkCudaError(Cuda_Recv(&pixel_start_id, 1, Cuda_INT, i, 0, Cuda_COMM_WORLD, &status));
            checkCudaError(Cuda_Recv(&pixel_end_id, 1, Cuda_INT, i, 1, Cuda_COMM_WORLD, &status));
            checkCudaError(Cuda_Recv(&temp[pixel_start_id], pixel_end_id - pixel_start_id + 1, Cuda_FLOAT, i, 2, Cuda_COMM_WORLD, &status));
        }
    }
    // Other processes send the data to the master
    else
    {
        checkCudaError(Cuda_Send(&pixel_start_id, 1, Cuda_INT, ROOT, 0, Cuda_COMM_WORLD));
        checkCudaError(Cuda_Send(&pixel_end_id,   1, Cuda_INT, ROOT, 1, Cuda_COMM_WORLD));
        checkCudaError(Cuda_Send(&temp[pixel_start_id], pixel_end_id - pixel_start_id + 1, Cuda_FLOAT, ROOT, 2, Cuda_COMM_WORLD));
    }
*/
    return temp;
}

/*
//-----------------------------------------------------
void CudaImage::workload(unsigned int aNumberOfElements,
                        unsigned int& aStartID,
                        unsigned int& aEndID) const
//-----------------------------------------------------
{
    int world_size;
    int rank;

    Cuda_Comm_size(Cuda_COMM_WORLD, &world_size);
    Cuda_Comm_rank(Cuda_COMM_WORLD, &rank);

    int last_element = -1;
    unsigned int element_per_task = aNumberOfElements / world_size;
    unsigned int remainder = aNumberOfElements % world_size;

    for (int i = 0; i < world_size; ++i)
    {
        aStartID = ++last_element;
        aEndID = last_element + element_per_task - 1;

        if (remainder > 0)
        {
            aEndID++;
            --remainder;
        }
        last_element = aEndID;

        // Exit the method
        if (rank == i) return;
    }
}
*/

//---------------------------------
int CudaImage::getNumberOfDevices()
//---------------------------------
{
    hipGetDeviceCount(&m_device_count);
    checkCudaError(__FILE__, __FUNCTION__, __LINE__);

    std::cerr << "Number of CUDA devices:\t" << m_device_count << std::endl;

    return m_device_count;
}


//-------------------------------------------------------
void CudaImage::checkCudaError(const char* aFileName,
                               const char* aFunctionName,
                               unsigned int aLineNumber)
//-------------------------------------------------------
{
	// Get the latest CUDA error.
	hipError_t error_code = hipGetLastError();

	// A CUDA error has occured.
	if (error_code != hipSuccess)
	{
        std::stringstream error_message;
        error_message << "CUDA error: " << hipGetErrorString(error_code) << std::endl;

        error_message << "\tin file: " << aFileName << std::endl;
        error_message << "\tin function: " << aFunctionName << std::endl;
        error_message << "\tat line: " << aLineNumber << std::endl;

		throw error_message;
	}
}


//-------------------------------------
void CudaImage::loadHost2Device() const
//-------------------------------------
{}


//-------------------------------------
void CudaImage::loadDevice2Host() const
//-------------------------------------
{}
