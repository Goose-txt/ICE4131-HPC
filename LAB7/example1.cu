
#include <hip/hip_runtime.h>
#include <iostream> // For cout and cerr
#include <cstdlib> // For rand(), srand(...) and RAND_MAX
#include <cmath> // For floor(...)
#include <ctime> // For time(NULL)

#include <hip/hip_runtime_api.h> // Main cuda header for high-level runtime
                              //   programming in C


//	Function declarations
__global__ void Kernel1(float* apOutputData,
                        float* apInputData0,
                        float* apInputData1);

void initializeArray(float* apArray);
void cleanup();
void checkCudaError(const char* aFileName,
                    const char* aFunctionName,
                    unsigned int aLineNumber);


// Constant variables
const unsigned int WIDTH    = 4096;

//	Global variables
float* g_p_host_float_set0   = NULL;
float* g_p_host_float_set1   = NULL;
float* g_p_host_float_set2   = NULL;
float* g_p_device_float_set0 = NULL;
float* g_p_device_float_set1 = NULL;
float* g_p_device_float_set2 = NULL;


//------------------------------------------------
__global__ void Kernel1(float* apOutputData,
                        float* apInputData0,
                        float* apInputData1,
                        int aWidth)
//------------------------------------------------
{
	// Get element index
	int i = blockDim.x * blockIdx.x + threadIdx.x;

    // i is a valid index
    if (i < aWidth)
    	// Element-wise sum
    	apOutputData[i] = apInputData0[i] + apInputData1[i];
}

int main()
{
	// Register an exit callback to make sure that the memory is released in cas
    // the program stops due to an error:
	atexit(cleanup);

	// Make sure there is at least one CUDA device
	int device_count;
	hipGetDeviceCount(&device_count);
	checkCudaError(__FILE__, __FUNCTION__, __LINE__);

	// There is no device
	if (!device_count)
	{
        std::cerr << "There is no CUDA device on this system. The program will terminate." << std::endl;
		return 1;
	}

    // Add the CUDA events
	hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    checkCudaError(__FILE__, __FUNCTION__, __LINE__);

    float host2device_memcpy_in_ms  = 0;
    float device2host_memcpy_in_ms  = 0;
    float kernel_execution_in_ms    = 0;

    // Compute the size of the array in number of bytes
	unsigned int array_size = WIDTH * sizeof(float);

	// Allocate the memory on host
    g_p_host_float_set0 = new float[WIDTH];
    g_p_host_float_set1 = new float[WIDTH];
    g_p_host_float_set2 = new float[WIDTH];

	// Allocate the memory on device
	hipMalloc((void**) &g_p_device_float_set0, array_size);
	hipMalloc((void**) &g_p_device_float_set1, array_size);
	hipMalloc((void**) &g_p_device_float_set2, array_size);
	checkCudaError(__FILE__, __FUNCTION__, __LINE__);

    // Initialize random number generator
	srand(time(NULL));

	// Initialize host memory using random numbers
	initializeArray(g_p_host_float_set0);
	initializeArray(g_p_host_float_set1);

    // Copy host memory to device memory
    hipEventRecord(start);
	hipMemcpy(g_p_device_float_set0, g_p_host_float_set0, array_size, hipMemcpyHostToDevice);
	hipMemcpy(g_p_device_float_set1, g_p_host_float_set1, array_size, hipMemcpyHostToDevice);
    hipEventRecord(stop);
    hipEventElapsedTime(&host2device_memcpy_in_ms, start, stop);
	checkCudaError(__FILE__, __FUNCTION__, __LINE__);

    // Configure the kernel
    int   DimBlock = 256; // 256 threads per block
    int   DimGrid  =  WIDTH / DimBlock;

    // Make sure there are enough blocks
    if (!(WIDTH % DimBlock)) ++DimGrid;

    // Run the kernel
    hipEventRecord(start);
	Kernel1<<< DimGrid, DimBlock >>>(g_p_device_float_set2,
		g_p_device_float_set0, g_p_device_float_set1, WIDTH);
	hipDeviceSynchronize();
    hipEventRecord(stop);
    hipEventElapsedTime(&kernel_execution_in_ms, start, stop);
	checkCudaError(__FILE__, __FUNCTION__, __LINE__);

    // Retrieve the result
    hipEventRecord(start);
    hipMemcpy(g_p_host_float_set2, g_p_device_float_set2, array_size, hipMemcpyDeviceToHost);
    hipEventRecord(stop);
    hipEventElapsedTime(&device2host_memcpy_in_ms, start, stop);
    checkCudaError(__FILE__, __FUNCTION__, __LINE__);

    // Destroy the CUDA events
    hipEventDestroy(start);
    hipEventDestroy(stop);

    // Display the results
    for (unsigned int i = 0; i < WIDTH; ++i)
    {
        printf("%i:\t%f + %f = %f\n",
            i,
            g_p_host_float_set0[i],
            g_p_host_float_set1[i],
            g_p_host_float_set2[i]);

    }

    // Display where time was spent
    float total_time = host2device_memcpy_in_ms + device2host_memcpy_in_ms + kernel_execution_in_ms;

    std::cout << "memcpy from host to device: " <<
        host2device_memcpy_in_ms << "ms. " <<
        100.0 * host2device_memcpy_in_ms / total_time << "% of total time" <<
        std::endl;

    std::cout << "memcpy from device to host: " <<
        device2host_memcpy_in_ms << "ms. " <<
        100.0 * device2host_memcpy_in_ms / total_time << "% of total time" <<
        std::endl;

    std::cout << "Kernel exectution: " <<
        kernel_execution_in_ms << "ms. " <<
        100.0 * kernel_execution_in_ms / total_time << "% of total time" <<
        std::endl;

    return 0;
}


//--------------------------------------------
void checkCudaError(const char* aFileName,
                    const char* aFunctionName,
                    unsigned int aLineNumber)
//--------------------------------------------
{
	// Get the latest CUDA error.
	hipError_t error_code = hipGetLastError();

	// A CUDA error has occured.
	if (error_code != hipSuccess)
	{
		// Display an error message in the console.
		fprintf(stderr, "CUDA error: %s\n\tin file: %s\n\tin function%s\n\tat line: %i.",
			hipGetErrorString(error_code),
			aFileName, aFunctionName, aLineNumber);

		// Quit the program.
		exit(EXIT_FAILURE);
	}
}


//------------
void cleanup()
//------------
{
	// Release memory
	printf( "Release memory if needed.\n");

	// Free host memory
    if (g_p_host_float_set0)
	{
		delete [] g_p_host_float_set0;
		g_p_host_float_set0 = NULL;
	}
	if (g_p_host_float_set1)
	{
		delete [] g_p_host_float_set1;
		g_p_host_float_set1 = NULL;
	}
	if (g_p_host_float_set2)
	{
		delete [] g_p_host_float_set2;
		g_p_host_float_set2 = NULL;
	}

    // Free device memory
    if (g_p_device_float_set0)
    {
        hipFree(g_p_device_float_set0);
        g_p_device_float_set0 = NULL;
    }
    if (g_p_device_float_set1)
    {
        hipFree(g_p_device_float_set1);
        g_p_device_float_set1 = NULL;
    }
    if (g_p_device_float_set2)
    {
        hipFree(g_p_device_float_set2);
        g_p_device_float_set2 = NULL;
    }
}


//----------------------------------
void initializeArray(float* apArray)
//----------------------------------
{
	for (unsigned int i = 0; i < WIDTH; ++i)
	{
		apArray[i] = floor(255.0f * (float)rand() / (float)RAND_MAX);
	}
}
