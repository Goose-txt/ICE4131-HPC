
#include <hip/hip_runtime.h>
#include <iostream>
#include <algorithm>

#include <hip/hip_runtime_api.h> // Main cuda header for high-level runtime
                              //   programming in C


//	Function declarations
__global__ void Kernel1(float* apOutputData,
                        float* apInputData0,
                        float* apInputData1);

void initializeArray(float* apArray);
void cleanup();
void checkCudaError(const char* aFileName,
                    const char* aFunctionName,
                    unsigned int aLineNumber);


// Constant variables
const unsigned int WIDTH    = 4096;

//	Global variables
float* g_p_host_float_set0   = NULL;
float* g_p_host_float_set1   = NULL;
float* g_p_host_float_set2   = NULL;
float* g_p_device_float_set0 = NULL;
float* g_p_device_float_set1 = NULL;
float* g_p_device_float_set2 = NULL;


int main()
{
	// Register an exit callback to make sure that the memory is released in cas
    // the program stops due to an error:
	atexit(cleanup);

	// Make sure there is at least one CUDA device
	int device_count;
	hipGetDeviceCount(&device_count);
	checkCudaError(__FILE__, __FUNCTION__, __LINE__);

	// There is no device
	if (!device_count)
	{
        std::cerr << "There is no CUDA device on this system. The program will terminate." << std::endl;
		return 1;
	}

    // Compute the size of the array in number of bytes
	unsigned int array_size = WIDTH * sizeof(float);

	// Allocate the memory on host
	g_p_host_float_set0 = (float*) malloc(array_size);
	g_p_host_float_set1 = (float*) malloc(array_size);
	g_p_host_float_set2 = (float*) malloc(array_size);

	// Allocate the memory on device
	hipMalloc((void**) &g_p_device_float_set0, array_size);
	hipMalloc((void**) &g_p_device_float_set1, array_size);
	hipMalloc((void**) &g_p_device_float_set2, array_size);
	checkCudaError(__FILE__, __FUNCTION__, __LINE__);

    // Initialize random number generator
	srand(time(NULL));

	// Initialize host memory using random numbers
	initializeArray(g_p_host_float_set0);
	initializeArray(g_p_host_float_set1);

    return 0;
}


//--------------------------------------------
void checkCudaError(const char* aFileName,
                    const char* aFunctionName,
                    unsigned int aLineNumber)
//--------------------------------------------
{
	// Get the latest CUDA error.
	hipError_t error_code = hipGetLastError();

	// A CUDA error has occured.
	if (error_code != hipSuccess)
	{
		// Display an error message in the console.
		fprintf(stderr, "CUDA error: %s\n\tin file: %s\n\tin function%s\n\tat line: %i.",
			hipGetErrorString(error_code),
			aFileName, aFunctionName, aLineNumber);

		// Quit the program.
		exit(EXIT_FAILURE);
	}
}


//------------
void cleanup()
//------------
{
	// Release memory
	printf( "Release memory if needed.\n");

	// Free host memory
    if (g_p_host_float_set0)
	{
		delete [] g_p_host_float_set0;
		g_p_host_float_set0 = NULL;
	}
	if (g_p_host_float_set1)
	{
		delete [] g_p_host_float_set1;
		g_p_host_float_set1 = NULL;
	}
	if (g_p_host_float_set2)
	{
		delete [] g_p_host_float_set2;
		g_p_host_float_set2 = NULL;
	}

    // Free device memory
    if (g_p_device_float_set0)
    {
        hipFree(g_p_device_float_set0);
        g_p_device_float_set0 = NULL;
    }
    if (g_p_device_float_set1)
    {
        hipFree(g_p_device_float_set1);
        g_p_device_float_set1 = NULL;
    }
    if (g_p_device_float_set2)
    {
        hipFree(g_p_device_float_set2);
        g_p_device_float_set2 = NULL;
    }
}


//----------------------------------
void initializeArray(float* apArray)
//----------------------------------
{
	for (unsigned int i = 0; i < WIDTH; ++i)
	{
		apArray[i] = floor(255.0f * (float)rand() / (float)RAND_MAX);
	}
}
